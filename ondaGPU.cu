#include "hip/hip_runtime.h"
/*
Desarrollado por Luis Yanes yanes.luis@gmail.com
Basado en codigo de NVIDIA Developer Toolkit
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include "SimParams.h"

#define k_blockDimX 16
#define k_blockDimY 16
#define k_blockSizeMin 64
#define RADIUS 4


__constant__ SimParams param;


__global__ void impulseKernel(float *p1,    // Presion salida (OUT)
                              const int dimx,
                              const int dimy,
                              const int dimz,
                              const int ixsource,
                              const int iysource,
                              const int izsource,
                              const float val)
{
  int gtidx = blockIdx.x * blockDim.x + threadIdx.x;    // Posicion en X
  int gtidy = blockIdx.y * blockDim.y + threadIdx.y;    // Posicion en Y
  if (gtidx == ixsource && gtidy == iysource){
    int stride_y    = dimx;
    int stride_z    = stride_y*dimy;

    int outIdx0 = (izsource ) * stride_z + (gtidy - 1)* stride_y + (gtidx - 1);
    int outIdx1 = (izsource ) * stride_z + (gtidy - 1)* stride_y + (gtidx );
    int outIdx2 = (izsource ) * stride_z + (gtidy )* stride_y + (gtidx - 1);
    int outIdx3 = (izsource ) * stride_z + (gtidy )* stride_y + (gtidx );
    int outIdx4 = (izsource - 1) * stride_z + (gtidy - 1)* stride_y + (gtidx - 1);
    int outIdx5 = (izsource - 1) * stride_z + (gtidy - 1)* stride_y + (gtidx );
    int outIdx6 = (izsource - 1) * stride_z + (gtidy )* stride_y + (gtidx - 1);
    int outIdx7 = (izsource - 1) * stride_z + (gtidy )* stride_y + (gtidx );
  
    p1[outIdx0] += val;
    p1[outIdx1] += val;
    p1[outIdx2] += val;
    p1[outIdx3] += val;
    p1[outIdx4] += val;
    p1[outIdx5] += val;
    p1[outIdx6] += val;
    p1[outIdx7] += val;
  }
}

__global__ void fofdKernel(float *p2,        // Presion salida   (OUT)
                           const float *p1,        // Presion actual   (IN OUT)
                           const float *p0,  // Presion anterior (IN)
                           const float *c,
                           const int dimx,
                           const int dimy,
                           const int dimz)
{
  //printf("Y = %d\n", blockIdx.y);
  bool valid = true;
  int gtidx = blockIdx.x * blockDim.x + threadIdx.x;    // Posicion en X
  int gtidy = blockIdx.y * blockDim.y + threadIdx.y;    // Posicion en Y
  int ltidx = threadIdx.x;
  int ltidy = threadIdx.y;
  int workx = blockDim.x;
  int worky = blockDim.y;
  __shared__ float tile[k_blockDimX + 2 * 1][k_blockDimY + 2 * 1];   // Bloque de trabajo    
  int stride_y    = dimx;
  int stride_z    = stride_y * dimy;
  int inputIndex  = gtidy * stride_y + gtidx;   // Posicion [x,y,0] transformada a una dimension para acceder al vector de entrada.
  int outputIndex = 0;

  float infront;
  float behind;
  float current;
  float tr1, tr2, tr3, tr4;
  float value = 0.0f;

	int tx = ltidx + 1;
	int ty = ltidy + 1;
	
  if (gtidx <= dimx-1 && gtidy <= dimy-1)
  {  
    if (gtidx < 1 || gtidx >= (dimx - 1))   // Out of bounds?
      valid = false;
    if (gtidy < 1 || gtidy >= (dimy - 1))   // Out of bounds?
      valid = false;

      // For simplicity we assume that the global size is equal to the actual
      // problem size; since the global size must be a multiple of the local size
      // this means the problem size must be a multiple of the local size (or
      // padded to meet this constraint).
      // Preload the "infront" and "behind" data
    behind = p1[inputIndex];    // Posiciones [x, y, z=0]
    inputIndex += stride_z;  // Z = 1
    
    current = p1[inputIndex];
    outputIndex = inputIndex;
    // Step through the xy-planes
    for (int z = 1 ; z < (dimz - 1) ; z++)
    {
      inputIndex += stride_z;   // Z = Z+1
      infront = p1[inputIndex];

      __syncthreads();          // Nos aseguramos que todos los hilos tengan los indices adecuados

      // Update the data slice in the local tile
      // Halo above & below
      if (ltidy < 1)
      {
        tile[ltidy][tx]             = p1[outputIndex - (1 * stride_y)];
        tile[ltidy + worky + 1][tx] = p1[outputIndex + (worky * stride_y)];
      }
      // Halo left & right
      if (ltidx < 1)
      {
        tile[ty][ltidx]             = p1[outputIndex - 1];
        tile[ty][ltidx + workx + 1] = p1[outputIndex + workx];
      }
      tile[ty][tx] = current;
          
      __syncthreads();  // Esperamos que todos los hilos hallan cargado la memoria compartida.

      tr1 = (tile[ty][tx-1] + tile[ty][tx+1] - 2.0f*current) / param.dx2;
      tr2 = (tile[ty-1][tx] + tile[ty+1][tx] - 2.0f*current) / param.dy2;
      tr3 = (infront + behind - 2.0f*current) / param.dz2;
      tr4 = pow(c[outputIndex], 2) * param.dt2 * (tr1 + tr2 + tr3);
      value = tr4 + (2.0f * p1[outputIndex]) - p0[outputIndex];
      // Store the output value
      if (! valid) value = current;
      p2[outputIndex] = value;
      
      behind = current;
      outputIndex = inputIndex;
      current = infront;
    }
  }
}


__global__ void bcKernel(float *p2,
                         float *p1,
                         const int dimx,
                         const int dimy,
                         const int dimz,
                         const int dampx,
                         const int dampy,
                         const int dampz)
{
  
  int gtidx = blockIdx.x * blockDim.x + threadIdx.x;    // Posicion en X
  int gtidy = blockIdx.y * blockDim.y + threadIdx.y;    // Posicion en Y

  int stride_y    = dimx;
  int stride_z    = stride_y * dimy;
  int inputIndex  = gtidy * stride_y + gtidx;   // Posicion [x,y,0] transformada a una dimension para acceder al vector de entrada.
  int bcIdx = 0;
  
  if (gtidx <= dimx-1 && gtidy <= dimy-1)
  {
    for (int z = 0 ; z < dimz ; z++)
    {
      bcIdx = z * stride_z + inputIndex;
      // X BC Width
      if (gtidx < dampx){   
        p2[bcIdx] *= exp(-pow(0.015*(gtidx),2));
        p1[bcIdx] *= exp(-pow(0.015*(gtidx),2));
      }

      if (gtidx > dimx - dampx){
        p2[bcIdx] *= exp(-pow(0.015*(dimx - gtidx),2));
        p1[bcIdx] *= exp(-pow(0.015*(dimx - gtidx),2));
      }
      
      // Y BC Heigth
      if (gtidy == 0){
        p2[bcIdx] = 0.0f;
        //p2[bcIdx] *= exp(-pow(0.015*(gtidy),2));
        //p1[bcIdx] *= exp(-pow(0.015*(gtidy),2));
      }
      
      if (gtidy > dimy - dampy){
        p2[bcIdx] *= exp(-pow(0.015*(dimy - gtidy),2));
        p1[bcIdx] *= exp(-pow(0.015*(dimy - gtidy),2));
      }
            
      // Z BC Depth
      // FREE SURFACE
      if (z < dampz){
        //p2[bcIdx] = 0.0f;
        p2[bcIdx] *= exp(-pow(0.015*(dampz - z),2));
        p1[bcIdx] *= exp(-pow(0.015*(dampz - z),2));
      }

      if (z > dimz - dampz){
        p2[bcIdx] *= exp(-pow(0.015*( dimz - dampz - z),2));
        p1[bcIdx] *= exp(-pow(0.015*( dimz - dampz - z),2));
      }

    }
  }
}

extern "C" void gpuGetNumDevices(int &v){
	int num;

	hipGetDeviceCount(&num);

	v = num;
}

extern "C" void gpuSetDevice(int d){
  hipSetDevice(d);
}

extern "C" void gpuMemcpyToDevice(float* dest, const float* src, unsigned int size)
{
    hipMemcpy(dest, src, size, hipMemcpyHostToDevice);
}

extern "C" void gpuMemcpyToHost(float* dest, const float* src, unsigned int size)
{
    hipMemcpy(dest, src, size, hipMemcpyDeviceToHost);
}

extern "C" void gpuMemcpyToSymbol(const void* src, unsigned int size)
{

    hipMemcpyToSymbol(HIP_SYMBOL(param), src, size);
}

extern "C" void* gpuMalloc(unsigned int size)
{
  void *pntr1;
  hipMalloc(&pntr1, size);
  return pntr1;
}

extern "C" void gpuImpulse(float* p1, const int dx, const int dy, const int dz, const int ix, const int iy, const int iz, const float val)
{
  dim3  dimBlock, dimGrid;

  dimBlock.x = k_blockDimX;
  dimBlock.y = k_blockDimY;
  dimGrid.x = (unsigned int) ceil( (float) dx / dimBlock.x );
  dimGrid.y = (unsigned int) ceil( (float) dy / dimBlock.y );
  
  hipDeviceSynchronize();
  impulseKernel<<<dimGrid, dimBlock>>>(p1, dx, dy, dz, ix, iy, iz, val);
}

extern "C" void gpuFirstOrderFD(float *p2, const float *p1, const float *p0, const float *c, const int dimx, const int dimy, const int dimz)
{
  dim3  dimBlock, dimGrid;

  dimBlock.x = k_blockDimX;
  dimBlock.y = k_blockDimY;
  dimGrid.x = (unsigned int) ceil( (float) dimx / dimBlock.x );
  dimGrid.y = (unsigned int) ceil( (float) dimy / dimBlock.y );
  
  hipDeviceSynchronize();
  fofdKernel<<<dimGrid, dimBlock>>>(p2, p1, p0, c, dimx, dimy, dimz);
}

extern "C" void gpuBoundaryConditions(float *p2, float *p1, const float *p0, const float *c, const int dimx, const int dimy, const int dimz, const int dampx, const int dampy, const int dampz)
{
  dim3  dimBlock, dimGrid;

  dimBlock.x = k_blockDimX;
  dimBlock.y = k_blockDimY;
  dimGrid.x = (unsigned int) ceil( (float) dimx / dimBlock.x );
  dimGrid.y = (unsigned int) ceil( (float) dimy / dimBlock.y );

  hipDeviceSynchronize();
  bcKernel<<<dimGrid, dimBlock>>>(p2, p1, dimx, dimy, dimz, dampx, dampy, dampz);

}

extern "C" void gpuFree(void* pntr)
{
  hipFree(pntr);
}
